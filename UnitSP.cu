#include "SpatialPooler.cu"
#include <assert.h>
#include <stdio.h>
#include <algorithm>

using namespace std;

template <typename T>
bool compare(const T* corr_vec, const T* out_vec, UInt size)
{
	for(int i=0; i < size; i++)
	{
		printf("%d \t %d \n", corr_vec[i], out_vec[i]);
		// printf("%d, ", out_vec[i]);
		if(corr_vec[i] != out_vec[i]) 
			return false;
	}
	return true;
}

void printErrorMessage(hipError_t error, int memorySize){
    printf("==================================================\n");
    printf("MEMORY ERROR  : %s\n", hipGetErrorString(error));
    printf("==================================================\n");
}


void setup_device2D(args& ar, bool* in_host, UInt* numPotential, UInt* potentialPools, Real* permanences, Real* boosts, const UInt SP_SIZE, const UInt IN_SIZE, const UInt MAX_CONNECTED)
{
    hipError_t result;
    // result = hipMalloc((void **) &ar_dev, sizeof(ar)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.in_dev, IN_SIZE*sizeof(bool)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.cols_dev, SP_SIZE*sizeof(bool)); if(result) printErrorMessage(result, 0);
	result = hipMalloc((void **) &ar.numPot_dev, SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0);
    result = hipMallocPitch((void **) &ar.pot_dev, &ar.pot_pitch_in_bytes, MAX_CONNECTED*sizeof(UInt), SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0); // width, height, x, y 
    result = hipMallocPitch((void **) &ar.per_dev, &ar.per_pitch_in_bytes, MAX_CONNECTED*sizeof(Real), SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMallocPitch((void **) &ar.odc_dev, &ar.odc_pitch_in_bytes, MAX_CONNECTED*sizeof(Real), SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMallocPitch((void **) &ar.adc_dev, &ar.adc_pitch_in_bytes, MAX_CONNECTED*sizeof(Real), SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMallocPitch((void **) &ar.boosts_dev, &ar.bst_pitch_in_bytes, MAX_CONNECTED*sizeof(Real), SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 

	// Memcpy to device
    // result = hipMemcpy(ar_dev, &ar, sizeof(ar), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.in_dev, in_host, IN_SIZE*sizeof(bool), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.numPot_dev, numPotential, SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy2D(ar.pot_dev, ar.pot_pitch_in_bytes, potentialPools, MAX_CONNECTED*sizeof(UInt), MAX_CONNECTED*sizeof(UInt), SP_SIZE, hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy2D(ar.per_dev, ar.per_pitch_in_bytes, permanences, MAX_CONNECTED*sizeof(Real), MAX_CONNECTED*sizeof(Real), SP_SIZE, hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy2D(ar.boosts_dev, ar.bst_pitch_in_bytes, boosts, MAX_CONNECTED*sizeof(Real), MAX_CONNECTED*sizeof(Real), SP_SIZE, hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
}

void setup_device1D(args& ar, bool* in_host, UInt* numPotential, UInt* potentialPools, Real* permanences, Real* boosts, const UInt SP_SIZE, const UInt IN_SIZE, const UInt MAX_CONNECTED)
{
    hipError_t result;
    // result = hipMalloc((void **) &ar_dev, sizeof(ar)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.in_dev, IN_SIZE*sizeof(bool)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.olaps_dev, SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0);
    // result = hipMalloc((void **) &ar.cols_dev, SP_SIZE*sizeof(bool)); if(result) printErrorMessage(result, 0);
	result = hipMalloc((void **) &ar.numPot_dev, SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0);
    result = hipMalloc((void **) &ar.pot_dev, MAX_CONNECTED*SP_SIZE*sizeof(UInt)); if(result) printErrorMessage(result, 0); // width, height, x, y 
    result = hipMalloc((void **) &ar.per_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.odc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.adc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 
    result = hipMalloc((void **) &ar.boosts_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)); if(result) printErrorMessage(result, 0); 

	// Memcpy to device
    // result = hipMemcpy(ar_dev, &ar, sizeof(ar), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.in_dev, in_host, IN_SIZE*sizeof(bool), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.numPot_dev, numPotential, SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.pot_dev, potentialPools, MAX_CONNECTED*SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.per_dev, permanences, MAX_CONNECTED*SP_SIZE*sizeof(Real), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    result = hipMemcpy(ar.boosts_dev, boosts, MAX_CONNECTED*SP_SIZE*sizeof(Real), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
}



void testCalculateOverlap()
{
	const UInt SP_SIZE = 16;
	const UInt IN_SIZE = 32;
	const UInt BLOCK_SIZE = 8;
	const UInt NUM_BLOCKS = SP_SIZE/BLOCK_SIZE;
	const UInt MAX_CONNECTED = 4;
	const UInt IN_BLOCK_SIZE = IN_SIZE/NUM_BLOCKS;
	Real threshold = 0.1;

							//0, 1, 2, 3, 4, 5, 6, 7
	bool in_host[IN_SIZE] =	{ 0, 1, 0, 1, 0, 1, 0, 1,
							//8, 9, 1, 1, 2, 3, 4, 5
		   		 	   		  1, 0, 1, 0, 1, 0, 1, 0,
   					   		  1, 1, 1, 1, 0, 0, 0, 0,
					   		  0, 0, 0, 0, 1, 1, 1, 1	
							};

	UInt potential[SP_SIZE*MAX_CONNECTED] = 	{ 0, 2, 3, 5,
   						 					  1, 3, 4, 7,
						 					  2, 5, 6, 7,
						 					  1, 4, 5, 11,
						 					  3, 10, 11, 15,
						 					  1, 9, 12, 14,
						 					  0, 13, 14, 15,
   						 					  1, 8, 9, 12, // 1st block
						 					  2, 5, 6, 7,
						 					  1, 4, 5, 6,
						 					  3, 4, 6, 7,
						 					  1, 11, 13, 14,
						 					  0, 8, 10, 15,
   						 					  1, 9, 10, 11,
						 					  2, 5, 9, 12,
						 					  1, 4, 5, 13, // 2nd block
											 };

	Real permanences[SP_SIZE*MAX_CONNECTED] = 	{ 0.09, 0.11, 0.09, 0.11, 
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
											  0.09, 0.11, 0.09, 0.11,
   											};

	Real boosts[SP_SIZE*MAX_CONNECTED];
	std::fill_n(boosts, SP_SIZE*MAX_CONNECTED, 1);

	UInt numPot[SP_SIZE];
	std::fill_n(numPot, SP_SIZE, MAX_CONNECTED);

	UInt correct_overlaps[SP_SIZE] = { 1, 2, 2, 0, 1, 1, 0, 2, 0, 0, 0, 1, 1, 0, 1, 1 }; 
	// std::fill_n(correct_overlaps, SP_SIZE, 0);

	UInt olaps[SP_SIZE];
	
	args ar;

	setup_device1D(ar, in_host, numPot, potential, permanences, boosts, SP_SIZE, IN_SIZE, MAX_CONNECTED);

	calculateOverlap_wrapper<<<NUM_BLOCKS, BLOCK_SIZE, BLOCK_SIZE*sizeof(UInt)>>>(ar.in_dev, ar.pot_dev, ar.per_dev, ar.boosts_dev, ar.numPot_dev, threshold, IN_BLOCK_SIZE, MAX_CONNECTED, ar.olaps_dev, SP_SIZE);

	hipError_t result = hipMemcpy(olaps, ar.olaps_dev, SP_SIZE*sizeof(UInt), hipMemcpyDeviceToHost); if(result) printErrorMessage(result, 0);

	assert(compare<UInt>(correct_overlaps, olaps, SP_SIZE));
	// compare<UInt>(correct_overlaps, olaps, SP_SIZE);
}

int main(int argc, const char * argv[])
{
	testCalculateOverlap();
}
