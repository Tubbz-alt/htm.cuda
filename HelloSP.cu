#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib>
#include <cmath>
#include <random>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>

#include "SpatialPooler.cu"

#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


using namespace std;

typedef unsigned int UInt;
typedef float Real;

UInt* generatePotentialPools(UInt* potentialPools, int cols, const UInt IN_BLOCK_SIZE, Real potentialPct, const UInt MAX_CONNECTED, UInt* numPotential)
{
    int connected = 0;
    for(int i=0; i < cols; i++)
    {
    	connected = 0;
		// Generated indeces should be in (0,IN_BLOCK_SIZE) and their count should be <= MAX_CONNECTED and around potentialPct*IN_BLOCK_SIZE
        for(int j=0; j < IN_BLOCK_SIZE; j++)
        {
            if((Real)(rand()%100)/100 <= potentialPct && connected < MAX_CONNECTED)
            {
                potentialPools[i*MAX_CONNECTED + connected++] = j; 
            }
        }
		numPotential[i] = connected;
    }
    return potentialPools;
}

Real initPermanencesConnected(Real synPermConnected_, Real synPermMax_)
{
	Real p = synPermConnected_ +
	             (synPermMax_ - synPermConnected_)*((Real)((rand()%100))/100);
	return p;
}

Real initPermanencesNotConnected(Real synPermConnected_)
{
	Real p = synPermConnected_ * (Real)((rand()%100))/100;
	return p;
}

Real* generatePermanences(Real* permanences, int cols, int inputSize, UInt* potential, Real connectedPct,
		Real synPermConnected_, Real synPermMax_, const UInt MAX_CONNECTED, UInt* numPotential,
	   	const UInt BLOCK_SIZE, const UInt IN_BLOCK_SIZE)
{
	int connected = 0;
	int curr_block = 0;
    bool found = false;

	for(int i=0; i < cols; i++)
	{
		connected = 0;
		// We need to only go through the input block corresponding to the current column
		// This means we need to convert current column to the input block number
		curr_block = floor(i / BLOCK_SIZE);
		// j is the global index of connection in the input matrix
		for(int j=curr_block*IN_BLOCK_SIZE; j < curr_block*IN_BLOCK_SIZE + IN_BLOCK_SIZE; j++)
		{
			// Find if this input is potentially connected with this column
			found=false;
            for(int k=0; k < numPotential[i]; k++)
            {
                if(potential[i*MAX_CONNECTED+k] == j % IN_BLOCK_SIZE) {
					found = true;
					break;
				}
            }
			// If there is, decide if it will be. The structure of the data is as follows:
		    // potential[col][index of the synapse on the segment] = index of input in the block
			// permanences[col][index of the synapse on the segment] = permanence of the synapse
            if(found)
            {
                if((Real)(rand()%100)/100 <= connectedPct)
                {
                    permanences[i*MAX_CONNECTED+connected++] = initPermanencesConnected(synPermConnected_, synPermMax_);
                }
                else
                {
                    permanences[i*MAX_CONNECTED+connected++] = initPermanencesNotConnected(synPermConnected_);
                }
            }
		}
	}
	return permanences;
}

// TO BE DELETED
// There should also be a parameter to raise permanences so that minimum number of synapses is connected.
UInt** computeConnected(Real** permanences, UInt** potential, UInt cols, UInt inputSize,
		Real synPermConnected_, const UInt MAX_CONNECTED, UInt* numPotential)
{
	UInt** connected_arr = new UInt*[cols];
	int connected = 0;
	for(int i=0; i < inputSize; i++)
	{
		connected = 0;
        connected_arr[i] = new UInt[MAX_CONNECTED];
		for(int j=0; j < numPotential[i]; j++)
		{
			if(permanences[i][j] < synPermConnected_)
			{
				connected_arr[i][connected++] = j;
			}
		}
	}
	return connected_arr;
}

bool* generate01(bool* ar, size_t size, Real inDensity)
{
	for(int i=0; i < size; i++)
	{
		ar[i] = (Real)(rand()%100)/100 <= inDensity ? 1 : 0;
	}
	return ar;
}

void visualize_input(bool* in_host, UInt* potentialPools, Real* permanences, UInt* numPotential, const UInt IN_SIZE, const UInt SP_SIZE, const UInt IN_BLOCK_SIZE, const UInt MAX_CONNECTED)
{
	printf("POTENTIAL CONNECTIONS WITH PERMANENCES\n");
	for(int i=0; i<SP_SIZE; i++)
	{
		for(int j=0; j<MAX_CONNECTED; j++)
			printf("%d \t", potentialPools[i*MAX_CONNECTED+j]);
		printf("\n");
		for(int j=0; j<numPotential[i]; j++)
			printf("%.2f\t", permanences[i*MAX_CONNECTED+j]);
		printf("\n");
		printf("%d \n", numPotential[i]);
	}

	printf("INPUT SDR\n");
	for(int i=0; i<IN_SIZE; i++)
	{
		printf("%d ", in_host[i]);
		if(i % IN_BLOCK_SIZE == 0 && i > 0)
			printf("\n");
	}
	printf("\n");
}

void visualize_output(bool* cols_host, const UInt SP_SIZE)
{
	// The final sparsity will approach target with increasing block size
	int ones = 0;
	for(int i=0; i < SP_SIZE; i++)
		if(cols_host[i] > 0) ones++;
	printf("Sparsity: %f \n", (Real)ones/SP_SIZE);
}

void printErrorMessage(hipError_t error, int memorySize){
    printf("==================================================\n");
    printf("MEMORY ERROR  : %s\n", hipGetErrorString(error));
    printf("==================================================\n");
}

int main(int argc, const char * argv[])
{
	srand(time(NULL));
	
	hiprandState dev_states;
	

    // construct input args
    args ar;
	ar.iteration_num=0;
	ar.learn=true;
	ar.localAreaDensity=0.02; // SP density after inhibition
    ar.potentialPct=0.5; // 
    ar.connectedPct=0.5;
    ar.stimulusThreshold=0;
    ar.synPermTrimThreshold=0.025;
    ar.synPermMax=1.0;
    ar.synPermConnected=0.1;
	ar.synPermActiveInc=0.05;
	ar.synPermInactiveDec=0.008;
	ar.synPermBelowStimulusInc=ar.synPermConnected / 10.0;
	ar.dutyCyclePeriod=1000;
	ar.boostStrength=0.05; // 0 means no boosting
	ar.minPctOdc=0.001;
	ar.update_period=50;
	ar.SP_SIZE = SP_SIZE;
	ar.MAX_CONNECTED = MAX_CONNECTED;
	ar.IN_BLOCK_SIZE = IN_BLOCK_SIZE;

	ar.num_connected = std::floor(MAX_CONNECTED*ar.connectedPct);

	// Host memory allocation
	size_t host_alloc_size = (IN_SIZE+SP_SIZE)*sizeof(bool);
    bool* cols_host = (bool*) malloc(host_alloc_size);
	bool* in_host = (bool*) &cols_host[SP_SIZE]; 

	// Host memory init	
	in_host = generate01(in_host, IN_SIZE, IN_DENSITY);

	// visualize_input(in_host, potentialPools, permanences, numPotential, IN_SIZE, SP_SIZE, IN_BLOCK_SIZE, MAX_CONNECTED);

	// Global memory pointers
	args* ar_dev;

	// Global memory allocation
    checkError( hipMalloc((void **) &ar_dev, sizeof(ar)) );

	checkError( hipMallocPitch((void **) &ar.pot_dev, &ar.pot_dev_pitch, ar.num_connected, SP_SIZE) );
	checkError( hipMallocPitch((void **) &ar.per_dev, &ar.per_dev_pitch, ar.num_connected, SP_SIZE) );
	checkError( hipMalloc((void **) &ar.boosts_dev, SP_SIZE*ar.num_connected*sizeof(Real)) );
    checkError( hipMalloc((void **) &ar.in_dev, IN_SIZE*sizeof(bool)) ); 
    checkError( hipMalloc((void **) &ar.olaps_dev, SP_SIZE*sizeof(UInt)) );
    checkError( hipMalloc((void **) &ar.cols_dev, SP_SIZE*sizeof(bool)) );
	checkError( hipMalloc((void **) &ar.numPot_dev, SP_SIZE*sizeof(UInt)) );
    checkError( hipMalloc((void **) &ar.odc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)) );
    checkError( hipMalloc((void **) &ar.adc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)) );
	checkError( hipMalloc((void **) &ar.minOdc_dev, NUM_BLOCKS*sizeof(Real)) );

	// Global memory initialization
	// Potential pools
	thrust::device_vector<UInt> input_indeces(IN_BLOCK_SIZE);
	thrust::sequence(input_indeces.begin(), input_indeces.end(), 0, 1);

	size_t sm = BLOCK_SIZE*sizeof(UInt);
	generatePotentialPools<<<SP_SIZE, BLOCK_SIZE, sm>>>(ar.pot_dev, ar.pot_dev_pitch, ar.num_connected, thrust::raw_pointer_cast(input_indeces.data()), &dev_states);

	// Permanences
	generatePermanences<<<SP_SIZE, ar.num_connected>>>(ar.per_dev, ar.per_dev_pitch, ar.connectedPct, ar.synPermConnected, ar.synPermMax, &dev_states);

	// Boosts
	thrust::device_ptr<float> dev_ptr(ar.boosts_dev);
	thrust::fill(dev_ptr, dev_ptr+SP_SIZE*ar.num_connected*sizeof(Real), 1.0);

	// Memcpy to device
    checkError( hipMemcpy(ar_dev, (void**) &ar, sizeof(ar), hipMemcpyHostToDevice) );
    checkError( hipMemcpy(ar.in_dev, in_host, IN_SIZE*sizeof(bool), hipMemcpyHostToDevice) );

	// Compute permanences
	sm = IN_BLOCK_SIZE*sizeof(UInt);
	calculateOverlap<<<SP_SIZE, BLOCK_SIZE, sm>>>(ar.in_dev, ar.pot_dev, ar.pot_dev_pitch, ar.per_dev, ar.per_dev_pitch, ar.boosts_dev, ar.synPermConnected, ar.num_connected)
	
	// Kernel call
	// sm = BLOCK_SIZE*(2*sizeof(Real) + sizeof(UInt)) + IN_BLOCK_SIZE*sizeof(bool);
    // compute<<<NUM_BLOCKS, BLOCK_SIZE, sm>>>(ar_dev, data_dev);

    // // Memcpy from device
    // checkError( hipMemcpy(cols_host, data_dev, SP_SIZE*sizeof(bool), hipMemcpyDeviceToHost)); 

	// visualize_output(cols_host, SP_SIZE);

    // hipFree(ar_dev); hipFree(data_dev);

    return 0;
}
