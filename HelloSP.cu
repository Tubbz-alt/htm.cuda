#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <ctime>
#include <cstdlib>
#include <cmath>
#include <random>
#include <assert.h>

#include "SpatialPooler.cu"

#define checkError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


using namespace std;

typedef unsigned int UInt;
typedef float Real;

UInt* generatePotentialPools(UInt* potentialPools, int cols, const UInt IN_BLOCK_SIZE, Real potentialPct, const UInt MAX_CONNECTED, UInt* numPotential)
{
    int connected = 0;
    for(int i=0; i < cols; i++)
    {
    	connected = 0;
		// Generated indeces should be in (0,IN_BLOCK_SIZE) and their count should be <= MAX_CONNECTED and around potentialPct*IN_BLOCK_SIZE
        for(int j=0; j < IN_BLOCK_SIZE; j++)
        {
            if((Real)(rand()%100)/100 <= potentialPct && connected < MAX_CONNECTED)
            {
                potentialPools[i*MAX_CONNECTED + connected++] = j; 
            }
        }
		numPotential[i] = connected;
    }
    return potentialPools;
}

Real initPermanencesConnected(Real synPermConnected_, Real synPermMax_)
{
	Real p = synPermConnected_ +
	             (synPermMax_ - synPermConnected_)*((Real)((rand()%100))/100);
	return p;
}

Real initPermanencesNotConnected(Real synPermConnected_)
{
	Real p = synPermConnected_ * (Real)((rand()%100))/100;
	return p;
}

Real* generatePermanences(Real* permanences, int cols, int inputSize, UInt* potential, Real connectedPct,
		Real synPermConnected_, Real synPermMax_, const UInt MAX_CONNECTED, UInt* numPotential,
	   	const UInt BLOCK_SIZE, const UInt IN_BLOCK_SIZE)
{
	int connected = 0;
	int curr_block = 0;
    bool found = false;

	for(int i=0; i < cols; i++)
	{
		connected = 0;
		// We need to only go through the input block corresponding to the current column
		// This means we need to convert current column to the input block number
		curr_block = floor(i / BLOCK_SIZE);
		// j is the global index of connection in the input matrix
		for(int j=curr_block*IN_BLOCK_SIZE; j < curr_block*IN_BLOCK_SIZE + IN_BLOCK_SIZE; j++)
		{
			// Find if this input is potentially connected with this column
			found=false;
            for(int k=0; k < numPotential[i]; k++)
            {
                if(potential[i*MAX_CONNECTED+k] == j % IN_BLOCK_SIZE) {
					found = true;
					break;
				}
            }
			// If there is, decide if it will be. The structure of the data is as follows:
		    // potential[col][index of the synapse on the segment] = index of input in the block
			// permanences[col][index of the synapse on the segment] = permanence of the synapse
            if(found)
            {
                if((Real)(rand()%100)/100 <= connectedPct)
                {
                    permanences[i*MAX_CONNECTED+connected++] = initPermanencesConnected(synPermConnected_, synPermMax_);
                }
                else
                {
                    permanences[i*MAX_CONNECTED+connected++] = initPermanencesNotConnected(synPermConnected_);
                }
            }
		}
	}
	return permanences;
}

// TO BE DELETED
// There should also be a parameter to raise permanences so that minimum number of synapses is connected.
UInt** computeConnected(Real** permanences, UInt** potential, UInt cols, UInt inputSize,
		Real synPermConnected_, const UInt MAX_CONNECTED, UInt* numPotential)
{
	UInt** connected_arr = new UInt*[cols];
	int connected = 0;
	for(int i=0; i < inputSize; i++)
	{
		connected = 0;
        connected_arr[i] = new UInt[MAX_CONNECTED];
		for(int j=0; j < numPotential[i]; j++)
		{
			if(permanences[i][j] < synPermConnected_)
			{
				connected_arr[i][connected++] = j;
			}
		}
	}
	return connected_arr;
}

bool* generate01(bool* ar, size_t size, Real inDensity)
{
	for(int i=0; i < size; i++)
	{
		ar[i] = (Real)(rand()%100)/100 <= inDensity ? 1 : 0;
	}
	return ar;
}

void visualize_input(bool* in_host, UInt* potentialPools, Real* permanences, UInt* numPotential, const UInt IN_SIZE, const UInt SP_SIZE, const UInt IN_BLOCK_SIZE, const UInt MAX_CONNECTED)
{
	printf("POTENTIAL CONNECTIONS WITH PERMANENCES\n");
	for(int i=0; i<SP_SIZE; i++)
	{
		for(int j=0; j<MAX_CONNECTED; j++)
			printf("%d \t", potentialPools[i*MAX_CONNECTED+j]);
		printf("\n");
		for(int j=0; j<numPotential[i]; j++)
			printf("%.2f\t", permanences[i*MAX_CONNECTED+j]);
		printf("\n");
		printf("%d \n", numPotential[i]);
	}

	printf("INPUT SDR\n");
	for(int i=0; i<IN_SIZE; i++)
	{
		printf("%d ", in_host[i]);
		if(i % IN_BLOCK_SIZE == 0 && i > 0)
			printf("\n");
	}
	printf("\n");
}

void visualize_output(bool* cols_host, const UInt SP_SIZE)
{
	// The final sparsity will approach target with increasing block size
	int ones = 0;
	for(int i=0; i < SP_SIZE; i++)
		if(cols_host[i] > 0) ones++;
	printf("Sparsity: %f \n", (Real)ones/SP_SIZE);
}

void printErrorMessage(hipError_t error, int memorySize){
    printf("==================================================\n");
    printf("MEMORY ERROR  : %s\n", hipGetErrorString(error));
    printf("==================================================\n");
}

int main(int argc, const char * argv[])
{
	srand(time(NULL));
	size_t sm = BLOCK_SIZE*(2*sizeof(Real) + sizeof(UInt)) + IN_BLOCK_SIZE*sizeof(bool);

    // construct input args
    args ar;
	ar.iteration_num=0;
	ar.learn=true;
	ar.localAreaDensity=0.02; // SP density after inhibition
    ar.potentialPct=0.5; // 
    ar.connectedPct=0.5;
    ar.stimulusThreshold=0;
    ar.synPermTrimThreshold=0.025;
    ar.synPermMax=1.0;
    ar.synPermConnected=0.1;
	ar.synPermActiveInc=0.05;
	ar.synPermInactiveDec=0.008;
	ar.synPermBelowStimulusInc=ar.synPermConnected / 10.0;
	ar.dutyCyclePeriod=1000;
	ar.boostStrength=0.05; // 0 means no boosting
	ar.minPctOdc=0.001;
	ar.update_period=50;
	ar.SP_SIZE = SP_SIZE;
	ar.MAX_CONNECTED = MAX_CONNECTED;
	ar.IN_BLOCK_SIZE = IN_BLOCK_SIZE;

	// Host memory pointers
    bool* cols_host; 									// = new bool[SP_SIZE];
	size_t host_alloc_size = IN_SIZE*sizeof(bool) + SP_SIZE*(sizeof(bool) + sizeof(UInt)) + SP_SIZE*MAX_CONNECTED*(sizeof(UInt) + 2*sizeof(Real));
	checkError( hipHostAlloc((void**) &cols_host, host_alloc_size, hipHostMallocDefault) );
	// result = hipHostAlloc((void**)&in_host, IN_SIZE*sizeof(bool), hipHostMallocDefault); if(result) printErrorMessage(result, 0);
	// result = hipHostAlloc((void**)&boosts, SP_SIZE*MAX_CONNECTED*sizeof(Real), hipHostMallocDefault); if(result) printErrorMessage(result, 0);
	// result = hipHostAlloc((void**)&potentialPools, SP_SIZE*MAX_CONNECTED*sizeof(UInt), hipHostMallocDefault); if(result) printErrorMessage(result, 0);
	// result = hipHostAlloc((void**)&permanences, SP_SIZE*MAX_CONNECTED*sizeof(Real), hipHostMallocDefault); if(result) printErrorMessage(result, 0);
	// result = hipHostAlloc((void**)&numPotential, SP_SIZE*sizeof(UInt), hipHostMallocDefault); if(result) printErrorMessage(result, 0);
	// result = hipHostAlloc((void**)&numConnected, SP_SIZE*sizeof(UInt), hipHostMallocDefault); if(result) printErrorMessage(result, 0);
	bool* in_host = (bool*) &cols_host[SP_SIZE]; 										// = new bool[IN_SIZE];
    UInt* potentialPools = (UInt*) &in_host[IN_SIZE];
	UInt* numPotential = &potentialPools[SP_SIZE*MAX_CONNECTED];									// = new UInt[SP_SIZE];
	// UInt* numConnected = &numPotential[SP_SIZE];									// = new UInt[SP_SIZE];
	Real* permanences = (Real*) &numPotential[SP_SIZE];
	Real* boosts = &permanences[SP_SIZE*MAX_CONNECTED];										// = new Real[SP_SIZE*MAX_CONNECTED];

	// Host memory allocation	
	memset(boosts, true, SP_SIZE*MAX_CONNECTED*sizeof(bool));
	memset(numPotential, 0, SP_SIZE*sizeof(UInt));
	// memset(numConnected, 0, SP_SIZE);

	potentialPools = generatePotentialPools(potentialPools, SP_SIZE, IN_BLOCK_SIZE, ar.potentialPct, MAX_CONNECTED, numPotential);
	permanences = generatePermanences(permanences, SP_SIZE, IN_SIZE, potentialPools, ar.connectedPct, ar.synPermConnected, ar.synPermMax, MAX_CONNECTED, numPotential,
					BLOCK_SIZE, IN_BLOCK_SIZE);
	in_host = generate01(in_host, IN_SIZE, IN_DENSITY);

	// visualize_input(in_host, potentialPools, permanences, numPotential, IN_SIZE, SP_SIZE, IN_BLOCK_SIZE, MAX_CONNECTED);

	// Global memory pointers
	args* ar_dev;
	void* data_dev;

	// Global memory allocation
	size_t device_alloc_size = host_alloc_size + SP_SIZE*sizeof(UInt) + 2*MAX_CONNECTED*SP_SIZE*sizeof(Real) + NUM_BLOCKS*sizeof(Real);
    checkError( hipMalloc((void **) &ar_dev, sizeof(ar)) );
	checkError( hipMalloc((void **) &data_dev, device_alloc_size) );
    // checkError( hipMalloc((void **) &ar.cols_dev, SP_SIZE*sizeof(bool)) );
    // checkError( hipMalloc((void **) &ar.in_dev, IN_SIZE*sizeof(bool)) ); 
    // checkError( hipMalloc((void **) &ar.boosts_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)) );
    // checkError( hipMalloc((void **) &ar.pot_dev, MAX_CONNECTED*SP_SIZE*sizeof(UInt)) );
    // checkError( hipMalloc((void **) &ar.per_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)) );
	// checkError( hipMalloc((void **) &ar.numPot_dev, SP_SIZE*sizeof(UInt)) );

    // checkError( hipMalloc((void **) &ar.olaps_dev, SP_SIZE*sizeof(UInt)) );
    // checkError( hipMalloc((void **) &ar.odc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)) );
    // checkError( hipMalloc((void **) &ar.adc_dev, MAX_CONNECTED*SP_SIZE*sizeof(Real)) );
    // checkError( hipMalloc((void **) &ar.minOdc_dev, NUM_BLOCKS*sizeof(Real)) );

	checkError( hipMemset(data_dev, 0, device_alloc_size) );

	// Memcpy to device
    checkError( hipMemcpy(ar_dev, (void**) &ar, sizeof(ar), hipMemcpyHostToDevice) );
    checkError( hipMemcpy(data_dev, cols_host, host_alloc_size, hipMemcpyHostToDevice) );
    // result = hipMemcpy(ar.in_dev, in_host, IN_SIZE*sizeof(bool), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    // result = hipMemcpy(ar.boosts_dev, boosts, MAX_CONNECTED*SP_SIZE*sizeof(Real), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    // result = hipMemcpy(ar.pot_dev, potentialPools, MAX_CONNECTED*SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    // result = hipMemcpy(ar.per_dev, permanences, MAX_CONNECTED*SP_SIZE*sizeof(Real), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);
    // result = hipMemcpy(ar.numPot_dev, numPotential, SP_SIZE*sizeof(UInt), hipMemcpyHostToDevice); if(result) printErrorMessage(result, 0);

	// Kernel call
    compute<<<NUM_BLOCKS, BLOCK_SIZE, sm>>>(ar_dev, data_dev);

    // Memcpy from device
    checkError( hipMemcpy(cols_host, data_dev, SP_SIZE*sizeof(bool), hipMemcpyDeviceToHost)); 

	visualize_output(cols_host, SP_SIZE);

    hipFree(ar_dev); hipFree(data_dev);

    return 0;
}
